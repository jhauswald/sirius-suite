#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <float.h>
#include <math.h>
#include <sys/time.h>
#include <pthread.h>
#include <string>

#include "../../utils/timer.h"

float feature_vect[] = {2.240018,    2.2570236,    0.11304555,   -0.21307051,
                        0.8988138,   0.039065503,  0.023874786,  0.13153112,
                        0.15324382,  0.16986738,   -0.020297153, -0.26773554,
                        0.40202165,  0.35923952,   0.060746543,  0.35402644,
                        0.086052455, -0.10499257,  0.04395058,   0.026407119,
                        -0.48301497, 0.120889395,  0.67980754,   -0.19875681,
                        -0.5443737,  -0.039534688, 0.20888293,   0.054865785,
                        -0.4846478,  0.1,          0.1,          0.1};

float *means_vect;
float *precs_vect;
float *weight_vect;
float *factor_vect;
float *score_vect;

__device__ __constant__ float logZero = -3.4028235E38;
__device__ __constant__ float maxLogValue = 7097004.5;
__device__ __constant__ float minLogValue = -7443538.0;
__device__ __constant__ float naturalLogBase = (float)1.00011595E-4;
__device__ __constant__ float inverseNaturalLogBase = 9998.841;
// fixed for a given accoustic model
__device__ __constant__ int comp_size = 32;
__device__ __constant__ int feat_size = 29;
__device__ __constant__ int senone_size = 5120;

extern "C"

    __global__ void
    computeScore(const float *feature_vect, float *means_vect,
                 float *precs_vect, float *weight_vect, float *factor_vect,
                 float *score_vect) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < senone_size) {
    float local_score_vect = logZero;

#pragma unroll 32
    for (int j = 0; j < comp_size; j++) {
      // getScore
      float logDval = 0.0f;
#pragma unroll 29
      for (int k = 0; k < feat_size; k++) {
        int idx = i + senone_size * j + k * comp_size * senone_size;
        float logDiff = feature_vect[k] - means_vect[idx];
        logDval += logDiff * logDiff * precs_vect[idx];
      }

      // Convert to the appropriate base.
      if (logDval != logZero) {
        logDval = logDval * inverseNaturalLogBase;
      }

      int idx2 = i + j * senone_size;

      // Add the precomputed factor, with the appropriate sign.
      logDval -= factor_vect[idx2];

      if (logDval < logZero) {
        logDval = logZero;
      }
      // end of getScore

      float logVal2 = logDval + weight_vect[idx2];

      float logHighestValue = local_score_vect;
      float logDifference = local_score_vect - logVal2;

      // difference is always a positive number
      if (logDifference < 0) {
        logHighestValue = logVal2;
        logDifference = -logDifference;
      }

      float logValue = -logDifference;
      float logInnerSummation;
      if (logValue < minLogValue) {
        logInnerSummation = 0.0;
      } else if (logValue > maxLogValue) {
        logInnerSummation = FLT_MAX;
      } else {
        if (logValue == logZero) {
          logValue = logZero;
        } else {
          logValue = logValue * naturalLogBase;
        }
        logInnerSummation = __expf(logValue);
      }

      logInnerSummation += 1.0;

      float returnLogValue;
      if (logInnerSummation <= 0.0) {
        returnLogValue = logZero;
      } else {
        returnLogValue = __logf(logInnerSummation) * inverseNaturalLogBase;
        if (returnLogValue > FLT_MAX) {
          returnLogValue = FLT_MAX;
        } else if (returnLogValue < -FLT_MAX) {
          returnLogValue = -FLT_MAX;
        }
      }
      // sum log
      local_score_vect = logHighestValue + returnLogValue;
    }
    score_vect[i] = local_score_vect;
  }
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    fprintf(stderr, "[ERROR] Invalid arguments provided.\n\n");
    fprintf(stderr, "Usage: %s [INPUT FILE]\n\n", argv[0]);
    exit(0);
  }
  STATS_INIT("kernel", "gpu_gaussian_mixture_model");
  PRINT_STAT_STRING("abrv", "gpu_gmm");

  float *dev_feat_vect;

  float cuda_elapsedTime;
  hipEvent_t eStart, eStop;
  int comp_size = 32;
  int senone_size = 5120;

  int means_array_size = senone_size * comp_size * comp_size;
  int comp_array_size = senone_size * comp_size;

  means_vect = (float *)malloc(means_array_size * sizeof(float));
  precs_vect = (float *)malloc(means_array_size * sizeof(float));
  weight_vect = (float *)malloc(comp_array_size * sizeof(float));
  factor_vect = (float *)malloc(comp_array_size * sizeof(float));

  float *means_vect2 = (float *)malloc(means_array_size * sizeof(float));
  float *precs_vect2 = (float *)malloc(means_array_size * sizeof(float));
  float *weight_vect2 = (float *)malloc(comp_array_size * sizeof(float));
  float *factor_vect2 = (float *)malloc(comp_array_size * sizeof(float));

  float *dev_means_vect;
  float *dev_precs_vect;
  float *dev_weight_vect;
  float *dev_factor_vect;

  score_vect = (float *)malloc(senone_size * sizeof(float));

  float *dev_score_vect;

  int blockSizeX = 256;
  int gridSizeX = (int)ceil(senone_size / blockSizeX);

  int div_grid = ((int)(gridSizeX / 32));
  gridSizeX = (div_grid + 1) * 32;

  // load model from file
  FILE *fp = fopen(argv[1], "r");
  if (fp == NULL) {  // checks for the file
    printf("\n Can’t open file");
    exit(-1);
  }

  int idx = 0;
  for (int i = 0; i < senone_size; i++) {
    for (int j = 0; j < comp_size; j++) {
      for (int k = 0; k < comp_size; k++) {
        float elem;
        fscanf(fp, "%f", &elem);
        means_vect[idx] = elem;
        idx = idx + 1;
      }
    }
  }

  idx = 0;
  for (int i = 0; i < senone_size; i++) {
    for (int j = 0; j < comp_size; j++) {
      for (int k = 0; k < comp_size; k++) {
        float elem;
        fscanf(fp, "%f", &elem);
        precs_vect[idx] = elem;
        idx = idx + 1;
      }
    }
  }

  idx = 0;
  for (int i = 0; i < senone_size; i++) {
    for (int j = 0; j < comp_size; j++) {
      float elem;
      fscanf(fp, "%f", &elem);
      weight_vect[idx] = elem;
      idx = idx + 1;
    }
  }

  idx = 0;
  for (int i = 0; i < senone_size; i++) {
    for (int j = 0; j < comp_size; j++) {
      float elem;
      fscanf(fp, "%f", &elem);
      factor_vect[idx] = elem;
      idx = idx + 1;
    }
  }

  fclose(fp);

  int idx3 = 0;
  for (int j = 0; j < comp_size; j++) {
    for (int i = 0; i < senone_size; i++) {
      int ij = j + i * comp_size;
      weight_vect2[idx3] = weight_vect[ij];
      factor_vect2[idx3] = factor_vect[ij];
      idx3 += 1;
    }
  }

  int idx4 = 0;
  for (int k = 0; k < comp_size; k++) {
    for (int j = 0; j < comp_size; j++) {
      for (int i = 0; i < senone_size; i++) {
        int ijk = k + comp_size * j + i * comp_size * comp_size;
        means_vect2[idx4] = means_vect[ijk];
        precs_vect2[idx4] = precs_vect[ijk];
        idx4 += 1;
      }
    }
  }

  for (int i = 0; i < senone_size; i++) {
    for (int j = 0; j < comp_size; j++) {
      for (int k = 0; k < 29; k++) {
        int ijk = k + comp_size * j + i * comp_size * comp_size;
        int kji = i + senone_size * j + k * comp_size * senone_size;
        if (means_vect2[kji] != means_vect[ijk]) {
          printf("%f != %f\n", means_vect2[kji], means_vect[ijk]);
        }
      }
    }
  }

  hipEventCreate(&eStart);
  hipEventCreate(&eStop);

  // just one time to load acoustic model
  hipMalloc((void **)&dev_means_vect, sizeof(float) * means_array_size);
  hipMalloc((void **)&dev_precs_vect, sizeof(float) * means_array_size);
  hipMalloc((void **)&dev_weight_vect, sizeof(float) * comp_array_size);
  hipMalloc((void **)&dev_factor_vect, sizeof(float) * comp_array_size);

  hipMemcpy(dev_means_vect, means_vect2, sizeof(float) * means_array_size,
             hipMemcpyHostToDevice);
  hipMemcpy(dev_precs_vect, precs_vect2, sizeof(float) * means_array_size,
             hipMemcpyHostToDevice);
  hipMemcpy(dev_weight_vect, weight_vect2, sizeof(float) * comp_array_size,
             hipMemcpyHostToDevice);
  hipMemcpy(dev_factor_vect, factor_vect2, sizeof(float) * comp_array_size,
             hipMemcpyHostToDevice);

  hipMalloc((void **)&dev_feat_vect, sizeof(float) * comp_size);
  hipMalloc((void **)&dev_score_vect, sizeof(float) * senone_size);

  PRINT_STAT_INT("blockSizeX", blockSizeX);
  PRINT_STAT_INT("gridSizeX", gridSizeX);

  dim3 block(128);
  dim3 grid;
  grid.x = (senone_size + block.x - 1) / block.x;

  if (grid.x < 32) grid.x = 32;

  hipEventRecord(eStart, 0);

  // each time needed for computing score of a given feature vect
  hipEventRecord(eStart, 0);
  hipMemcpy(dev_feat_vect, feature_vect, comp_size * sizeof(float),
             hipMemcpyHostToDevice);
  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);
  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);
  PRINT_STAT_DOUBLE("host_to_device", cuda_elapsedTime);

  hipEventRecord(eStart, 0);
  computeScore << <grid, block>>> (dev_feat_vect, dev_means_vect,
                                   dev_precs_vect, dev_weight_vect,
                                   dev_factor_vect, dev_score_vect);

  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);

  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);
  PRINT_STAT_DOUBLE("gpu_gmm", cuda_elapsedTime);

  hipEventRecord(eStart, 0);
  hipMemcpy(score_vect, dev_score_vect, senone_size * sizeof(float),
             hipMemcpyDeviceToHost);
  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);
  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);
  PRINT_STAT_DOUBLE("device_to_host", cuda_elapsedTime);

  STATS_END();

#if TESTING
  FILE *f = fopen("../input/gmm_scoring.gpu", "w");

  for (int i = 0; i < senone_size; ++i) fprintf(f, "%.0f\n", score_vect[i]);

  fclose(f);
#endif

  hipEventRecord(eStop, 0);
  hipEventSynchronize(eStop);

  hipEventElapsedTime(&cuda_elapsedTime, eStart, eStop);

  free(means_vect);
  free(precs_vect);

  free(weight_vect);
  free(factor_vect);

  free(score_vect);

  hipFree(dev_means_vect);
  hipFree(dev_precs_vect);
  hipFree(dev_weight_vect);
  hipFree(dev_factor_vect);

  hipFree(dev_feat_vect);
  hipFree(dev_score_vect);
}
